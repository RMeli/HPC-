#include "hip/hip_runtime.h"
#include "mm_cu.h"

#include <stdexcept>

namespace cuda
{
    __global__ void mm_kernel(const double *a, const double *b, double *c, size_t n_rows, size_t n, size_t n_cols)
    {
        auto i = blockIdx.x * blockDim.x + threadIdx.x;
        auto j = blockIdx.y * blockDim.y + threadIdx.y;

        if (i < n_rows && j < n_cols)
        {
            double r = 0.0;
            for (size_t k = 0; k < n; k++)
            {
                r += a[index(i, k, n)] * b[index(k, j, n_cols)];
            }

            c[index(i, j, n_cols)] = r;
        }
    }

    auto mm(const double *a,
            const double *b,
            double *c,
            size_t n_rows, size_t n, size_t n_cols,
            size_t n_threads_per_block, size_t n_blocks) -> void
    {
        double *a_device;
        double *b_device;
        double *c_device;

        auto a_size_bytes = sizeof(double) * n_rows * n;
        auto b_size_bytes = sizeof(double) * n * n_cols;
        auto c_size_bytes = sizeof(double) * n_rows * n_cols;

        hipMalloc(&a_device, a_size_bytes);
        hipMalloc(&b_device, b_size_bytes);
        hipMalloc(&c_device, c_size_bytes);

        hipMemcpy(a_device, a, a_size_bytes, hipMemcpyHostToDevice);
        hipMemcpy(b_device, b, b_size_bytes, hipMemcpyHostToDevice);

        dim3 threads_per_block(n_threads_per_block, n_threads_per_block);
        dim3 blocks(n_blocks, n_blocks);

        mm_kernel<<<threads_per_block, blocks>>>(a_device, b_device, c_device, n_rows, n, n_cols);
        auto err = hipGetLastError();
        if (hipSuccess != err)
        {
            throw std::runtime_error(hipGetErrorString(err));
        }

        hipMemcpy(c, c_device, c_size_bytes, hipMemcpyDeviceToHost);

        hipFree(a_device);
        hipFree(b_device);
        hipFree(c_device);
    }
}
